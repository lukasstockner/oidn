// Copyright 2022 Intel Corporation
// SPDX-License-Identifier: Apache-2.0

#include "cuda_engine.h"
#include "cuda_external_buffer.h"
#include "cuda_conv.h"
#include "../gpu/gpu_autoexposure.h"
#include "../gpu/gpu_input_process.h"
#include "../gpu/gpu_output_process.h"
#include "../gpu/gpu_error_process.h"
#include "../gpu/gpu_pool.h"
#include "../gpu/gpu_upsample.h"
#include "../gpu/gpu_image_copy.h"
#include "../gpu/gpu_tensor_copy.h"

OIDN_NAMESPACE_BEGIN

  CUDAEngine::CUDAEngine(const Ref<CUDADevice>& device,
                         hipStream_t stream)
    : device(device.get()),
      stream(stream) {}

  Ref<Buffer> CUDAEngine::newExternalBuffer(ExternalMemoryTypeFlag fdType,
                                            int fd, size_t byteSize)
  {
    return makeRef<CUDAExternalBuffer>(this, fdType, fd, byteSize);
  }

  Ref<Buffer> CUDAEngine::newExternalBuffer(ExternalMemoryTypeFlag handleType,
                                            void* handle, const void* name, size_t byteSize)
  {
    return makeRef<CUDAExternalBuffer>(this, handleType, handle, name, byteSize);
  }

  bool CUDAEngine::isSupported(const TensorDesc& desc) const
  {
    // CUTLASS stores tensor strides in 32-bit signed integers
    return Engine::isSupported(desc) && desc.getNumElements() <= INT32_MAX;
  }

  std::shared_ptr<Conv> CUDAEngine::newConv(const ConvDesc& desc)
  {
    return newCUDAConv(this, desc);
  }

  std::shared_ptr<Pool> CUDAEngine::newPool(const PoolDesc& desc)
  {
    return std::make_shared<GPUPool<CUDAEngine, half, TensorLayout::hwc>>(this, desc);
  }

  std::shared_ptr<Upsample> CUDAEngine::newUpsample(const UpsampleDesc& desc)
  {
    return std::make_shared<GPUUpsample<CUDAEngine, half, TensorLayout::hwc>>(this, desc);
  }

  std::shared_ptr<TensorCopy> CUDAEngine::newTensorCopy(const TensorCopyDesc& desc)
  {
    return std::make_shared<GPUTensorCopy<CUDAEngine, half, TensorLayout::hwc>>(this, desc);
  }

  std::shared_ptr<Autoexposure> CUDAEngine::newAutoexposure(const ImageDesc& srcDesc)
  {
    return std::make_shared<GPUAutoexposure<CUDAEngine, 1024>>(this, srcDesc);
  }

  std::shared_ptr<InputProcess> CUDAEngine::newInputProcess(const InputProcessDesc& desc)
  {
    return std::make_shared<GPUInputProcess<CUDAEngine, half, TensorLayout::hwc>>(this, desc);
  }

  std::shared_ptr<OutputProcess> CUDAEngine::newOutputProcess(const OutputProcessDesc& desc)
  {
    return std::make_shared<GPUOutputProcess<CUDAEngine, half, TensorLayout::hwc>>(this, desc);
  }

  std::shared_ptr<ErrorProcess> CUDAEngine::newErrorProcess(const ErrorProcessDesc& desc)
  {
    return std::make_shared<GPUErrorProcess<CUDAEngine, half, TensorLayout::hwc>>(this, desc);
  }

  std::shared_ptr<ImageCopy> CUDAEngine::newImageCopy()
  {
    return std::make_shared<GPUImageCopy<CUDAEngine>>(this);
  }

  void* CUDAEngine::usmAlloc(size_t byteSize, Storage storage)
  {
    if (byteSize == 0)
      return nullptr;

    void* ptr = nullptr;

    switch (storage)
    {
    case Storage::Host:
      checkError(hipHostMalloc(&ptr, byteSize));
      return ptr;

    case Storage::Device:
      checkError(hipMalloc(&ptr, byteSize));
      return ptr;

    case Storage::Managed:
      checkError(hipMallocManaged(&ptr, byteSize));
      return ptr;

    default:
      throw Exception(Error::InvalidArgument, "invalid storage mode");
    }
  }

  void CUDAEngine::usmFree(void* ptr, Storage storage)
  {
    if (ptr == nullptr)
      return;

    if (storage == Storage::Host)
      checkError(hipHostFree(ptr));
    else
      checkError(hipFree(ptr));
  }

  void CUDAEngine::usmCopy(void* dstPtr, const void* srcPtr, size_t byteSize)
  {
    checkError(hipMemcpy(dstPtr, srcPtr, byteSize, hipMemcpyDefault));
  }

  void CUDAEngine::submitUSMCopy(void* dstPtr, const void* srcPtr, size_t byteSize)
  {
    checkError(hipMemcpyAsync(dstPtr, srcPtr, byteSize, hipMemcpyDefault, stream));
  }

  namespace
  {
    void CUDART_CB hostFuncCallback(hipStream_t stream, hipError_t status, void* fPtr)
    {
      std::unique_ptr<std::function<void()>> f(reinterpret_cast<std::function<void()>*>(fPtr));
      if (status == hipSuccess)
        (*f)();
    }
  }

  void CUDAEngine::submitHostFunc(std::function<void()>&& f)
  {
    auto fPtr = new std::function<void()>(std::move(f));
    checkError(hipStreamAddCallback(stream, hostFuncCallback, fPtr, 0));
  }

  void CUDAEngine::wait()
  {
    checkError(hipStreamSynchronize(stream));
  }

OIDN_NAMESPACE_END